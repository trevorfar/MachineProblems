#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#define TOLERANCE 0.0001

const int test_sizes[] = { 256, 512, 1024, 2048, 4096 };
const int tile_sizes[] = { 2, 4, 8, 16, 32 };
const int num_tests = sizeof(test_sizes) / sizeof(test_sizes[0]);
const int num_tiles = sizeof(tile_sizes) / sizeof(tile_sizes[0]);

__global__ void matrixMulTiled(float* P, float* M, float* N, int width, int TILE_WIDTH) {
    extern __shared__ float sharedMemory[];
    float* Mds = sharedMemory;
    float* Nds = &sharedMemory[TILE_WIDTH * TILE_WIDTH];

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_WIDTH + ty;
    int col = blockIdx.x * TILE_WIDTH + tx;
    float Pvalue = 0.0;

    for (int ph = 0; ph < width / TILE_WIDTH; ++ph) {
        if (row < width && (ph * TILE_WIDTH + tx) < width)
            Mds[ty * TILE_WIDTH + tx] = M[row * width + ph * TILE_WIDTH + tx];
        else
            Mds[ty * TILE_WIDTH + tx] = 0.0;

        if (col < width && (ph * TILE_WIDTH + ty) < width)
            Nds[ty * TILE_WIDTH + tx] = N[(ph * TILE_WIDTH + ty) * width + col];
        else
            Nds[ty * TILE_WIDTH + tx] = 0.0;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += Mds[ty * TILE_WIDTH + k] * Nds[k * TILE_WIDTH + tx];

        __syncthreads();
    }

    if (row < width && col < width)
        P[row * width + col] = Pvalue;
}

void verifyMatrix(float* matrix1, float* matrix2, int numDimensions) {
    for (int i = 0; i < numDimensions * numDimensions; i++) {
        if (fabs(matrix1[i] - matrix2[i]) > TOLERANCE) {
            printf("Test FAILED\n");
            return;
        }
    }
    printf("Test PASSED\n");
}

void hostFunction(int width, int TILE_WIDTH) {
    size_t size = width * width * sizeof(float);
    float* h_M, * h_N, * h_P, * d_M, * d_N, * d_P;

    h_M = (float*)malloc(size);
    h_N = (float*)malloc(size);
    h_P = (float*)malloc(size);
    float* h_P_cpu = (float*)malloc(size);

    for (int i = 0; i < width * width; i++) {
        h_M[i] = static_cast<float>(rand() % 10);
        h_N[i] = static_cast<float>(rand() % 10);
    }

    hipMalloc((void**)&d_M, size);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_P, size);

    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

    dim3 dimGrid((width + TILE_WIDTH - 1) / TILE_WIDTH, (width + TILE_WIDTH - 1) / TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    size_t sharedMemSize = 2 * TILE_WIDTH * TILE_WIDTH * sizeof(float);

    matrixMulTiled << <dimGrid, dimBlock, sharedMemSize >> > (d_P, d_M, d_N, width, TILE_WIDTH);

    hipMemcpy(h_P_cpu, d_P, size, hipMemcpyDeviceToHost);
    verifyMatrix(h_P_cpu, h_P_cpu, width);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
    free(h_M);
    free(h_N);
    free(h_P);
    free(h_P_cpu);
}

int main() {
    for (int i = 0; i < num_tests; i++) {
        int matrix_size = test_sizes[i];
        for (int j = 0; j < num_tiles; j++) {
            int TILE_WIDTH = tile_sizes[j];
            printf("Running test for matrix size: %d x %d with TILE_WIDTH: %d\n", matrix_size, matrix_size, TILE_WIDTH);

            float gpu_time = 0.0f;
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipDeviceSynchronize();

            hipEventRecord(start, 0);
            hostFunction(matrix_size, TILE_WIDTH);
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&gpu_time, start, stop);

            printf("GPU Execution Time for %d x %d with TILE_WIDTH %d: %f ms\n\n", matrix_size, matrix_size, TILE_WIDTH, gpu_time);

            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
    }
    return 0;
}
